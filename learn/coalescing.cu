#include "hip/hip_runtime.h"
#include <cstdio>
#include <cassert>
#include "common.hu"

template <typename T>
__global__ void offset(T* a, int s) {
  int i = blockDim.x * blockIdx.x + threadIdx.x + s;
  a[i] = a[i] + 1;
}

template <typename T>
__global__ void stride(T* a, int s) {
  int i = (blockDim.x * blockIdx.x + threadIdx.x) * s;
  a[i] = a[i] + 1;
}

template <typename T>
void runTest(int deviceId, int nMB) {
  int blockSize = 256;
  float ms;

  T *d_a;
  hipEvent_t startEvent, stopEvent;
    
  int n = nMB*1024*1024/sizeof(T);
  CUDA_CHECK( hipMalloc(&d_a, n * 33 * sizeof(T)) );

  CUDA_CHECK( hipEventCreate(&startEvent) );
  CUDA_CHECK( hipEventCreate(&stopEvent) );

  printf("Offset (GB/s):\n");
  offset<<<n/blockSize, blockSize>>>(d_a, 0);

  for (int i = 0; i <= 32; i++) {
    CUDA_CHECK( hipMemset(d_a, 0, n * sizeof(T)) );

    CUDA_CHECK( hipEventRecord(startEvent,0) );
    offset<<<n/blockSize, blockSize>>>(d_a, i);
    CUDA_CHECK( hipEventRecord(stopEvent,0) );
    CUDA_CHECK( hipEventSynchronize(stopEvent) );

    CUDA_CHECK( hipEventElapsedTime(&ms, startEvent, stopEvent) );
    printf("%f ", i, 2*nMB/ms);
  }

  printf("\n");
  printf("Stride (GB/s):\n");

  stride<<<n/blockSize, blockSize>>>(d_a, 1); // warm up
  for (int i = 1; i <= 32; i++) {
    CUDA_CHECK( hipMemset(d_a, 0, n * sizeof(T)) );

    CUDA_CHECK( hipEventRecord(startEvent,0) );
    stride<<<n/blockSize, blockSize>>>(d_a, i);
    CUDA_CHECK( hipEventRecord(stopEvent,0) );
    CUDA_CHECK( hipEventSynchronize(stopEvent) );

    CUDA_CHECK( hipEventElapsedTime(&ms, startEvent, stopEvent) );
    printf("%f ", i, 2*nMB/ms);
  }

  CUDA_CHECK( hipEventDestroy(startEvent) );
  CUDA_CHECK( hipEventDestroy(stopEvent) );
  hipFree(d_a);
  printf("\n");
}

int main(int argc, char **argv)
{
  int nMB = 4;
  int deviceId = 0;
  bool bFp64 = false;

  for (int i = 1; i < argc; i++) {    
    if (!strncmp(argv[i], "dev=", 4))
      deviceId = atoi((char*)(&argv[i][4]));
    else if (!strcmp(argv[i], "fp64"))
      bFp64 = true;
  }
  
  hipDeviceProp_t prop;
  
  CUDA_CHECK(hipSetDevice(deviceId));
  CUDA_CHECK(hipGetDeviceProperties(&prop, deviceId));
  printf("Device: %s\n", prop.name);
  printf("Transfer size (MB): %d\n", nMB);
  
  printf("%s Precision\n", bFp64 ? "Double" : "Single");
  
  if (bFp64) runTest<double>(deviceId, nMB);
  else       runTest<float>(deviceId, nMB);
}